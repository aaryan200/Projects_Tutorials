#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 2048 * 2048 // Number of elements in each vector

/*
 * Optimize this already-accelerated codebase. Work iteratively,
 * and use nsys to support your work.
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 300us.
 *
 * Some bugs have been placed in this codebase for your edification.
 */

__global__ void initWith(int val, int* arr) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < N) arr[tid] = val;
}

__global__ void saxpy(int *a, int *b, int *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < N) c[tid] = 2 * a[tid] + b[tid];
}

int main()
{
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    int threads_per_block = 512;
    int number_of_blocks = (N + threads_per_block - 1)/ threads_per_block;

    int *a, *b, *c;

    int size = N * sizeof(int); // The total number of bytes per vector

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    hipMemPrefetchAsync(a, size, deviceId);
    hipMemPrefetchAsync(b, size, deviceId);
    hipMemPrefetchAsync(c, size, deviceId);

    initWith<<<number_of_blocks, threads_per_block>>>(2, a);
    initWith<<<number_of_blocks, threads_per_block>>>(1, b);
    initWith<<<number_of_blocks, threads_per_block>>>(0, a);

    // Initialize memory
    // for (int i = 0; i < N; ++i)
    // {
    //     a[i] = 2;
    //     b[i] = 1;
    //     c[i] = 0;
    // }

    saxpy<<<number_of_blocks, threads_per_block>>>(a, b, c);
    hipDeviceSynchronize();

    hipMemPrefetchAsync(c, size, hipCpuDeviceId);

    // Print out the first and last 5 values of c for a quality check
    for (int i = 0; i < 5; ++i) printf("c[%d] = %d, ", i, c[i]);
    printf("\n");

    for (int i = N - 5; i < N; ++i) printf("c[%d] = %d, ", i, c[i]);
    printf("\n");

    hipFree(a);
    hipFree(b);
    hipFree(c);
}
