
#include <hip/hip_runtime.h>
#include <stdio.h>

void helloCPU()
{
  printf("Hello from the CPU.\n");
}

/*
 * Refactor the `helloGPU` definition to be a kernel
 * that can be launched on the GPU. Update its message
 * to read "Hello from the GPU!"
 */

__global__ void helloGPU()
{
  printf("Hello also from the GPU.\n");
}

int main()
{

  helloCPU();

  /*
   * Refactor this call to `helloGPU` so that it launches
   * as a kernel on the GPU.
   */

  helloGPU<<<1, 1>>>();

  /*
   * Add code below to synchronize on the completion of the
   * `helloGPU` kernel completion before continuing the CPU
   * thread.
   */
   hipDeviceSynchronize();
}
