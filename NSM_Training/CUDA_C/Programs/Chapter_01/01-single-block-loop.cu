
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

__global__ void loop(int N)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int bdim = blockIdx.x;
  int global_id = bid * bdim + tid;
  printf("Block %d, Thread %d, Global: %d\n", tid, bid, global_id);
}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, only use 1 block of threads.
   */

  int N = 10;
  loop<<<1, N>>>(N);
  hipDeviceSynchronize();
}
