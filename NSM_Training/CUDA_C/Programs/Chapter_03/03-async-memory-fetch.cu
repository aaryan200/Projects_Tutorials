
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void initWith(float num, float *a, int N)
{

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;
  float *h_c;

  hipMalloc(&a, size);
  hipMalloc(&b, size);
  hipMalloc(&c, size);
  hipHostMalloc(&h_c, size, hipHostMallocDefault);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;
  numberOfBlocks = 32 * numberOfSMs;

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  /*
   * Create 3 streams to run initialize the 3 data vectors in parallel.
   */

  hipStream_t stream1, stream2, stream3;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  hipStreamCreate(&stream3);

  /*
   * Give each `initWith` launch its own non-standard stream.
   */

  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream1>>>(3, a, N);
  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream2>>>(4, b, N);
  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream3>>>(0, c, N);
    
  /*
   * Synchronize - we are using different non-standard streams for initialisation
   * and calcs, so we need to explicitly synchronize to ensure all data has been
   * initialised before starting the calculations 
   */
  hipError_t asyncErrInit = hipDeviceSynchronize();
  if(asyncErrInit != hipSuccess) printf("Error init: %s\n", hipGetErrorString(asyncErrInit));
  
  int segmentSize = size / 4;
  int segLen = N / 4;
    
  for (int i = 0; i < 4; i++) {
      hipStream_t stream;
      hipStreamCreate(&stream);
      
      int segOffset = i * segLen;
      
      // Execute the add vectors kernel on this non-default stream
      addVectorsInto<<<numberOfBlocks, threadsPerBlock, 0, stream>>>(&c[segOffset], // provide the pointer to this location
                                                                     &a[segOffset],
                                                                     &b[segOffset],
                                                                     segLen);
      
      // Use cudaMemcpyAsync to copy this segment to host over the same non-default stream
      hipMemcpyAsync(&h_c[segOffset],
                      &c[segOffset],
                      segmentSize,
                      hipMemcpyDeviceToHost,
                      stream);
      
      hipStreamDestroy(stream);
  }

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  checkElementsAre(7, h_c, N);

  /*
   * Destroy streams when they are no longer needed.
   */

  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipStreamDestroy(stream3);

  hipFree(a);
  hipFree(b);
  hipFree(c);
  hipHostFree(h_c);
}
