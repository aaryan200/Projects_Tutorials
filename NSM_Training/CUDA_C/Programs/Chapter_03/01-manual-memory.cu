
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void initWith(float num, float *a, int N)
{

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *device_c, *host_c;

  hipMalloc(&a, size);
  hipMalloc(&b, size);
  hipMalloc(&device_c, size);

  hipHostMalloc(&host_c, size, hipHostMallocDefault);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;
  numberOfBlocks = 32 * numberOfSMs;

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  /*
   * Create 3 streams to run initialize the 3 data vectors in parallel.
   */

  hipStream_t stream1, stream2, stream3;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  hipStreamCreate(&stream3);

  /*
   * Give each `initWith` launch its own non-standard stream.
   */

  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream1>>>(3, a, N);
  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream2>>>(4, b, N);
  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream3>>>(0, device_c, N);

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(device_c, a, b, N);

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Add vectors error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Async error: %s\n", hipGetErrorString(asyncErr));

  hipMemcpy(host_c, device_c, size, hipMemcpyDeviceToHost);

  checkElementsAre(7, host_c, N);

  /*
   * Destroy streams when they are no longer needed.
   */

  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipStreamDestroy(stream3);

  hipFree(a);
  hipFree(b);
  hipFree(device_c);
  hipHostFree(host_c);
}
